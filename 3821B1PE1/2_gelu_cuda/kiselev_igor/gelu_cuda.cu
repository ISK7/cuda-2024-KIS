#include "hip/hip_runtime.h"
// Copyright (c) 2024 Kiselev Igor
#include "gelu_cuda.h"

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cmath>

__global__ void geluKernel(const float* input, float* result, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        float x = input[idx];
        result[idx] = 0.5f * x * (1.0f + tanh(sqrt(2.0f / M_PI) * (x + 0.044715f * x * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    size_t vectorSize = input.size();
    std::vector<float> result(vectorSize);
    size_t sizeInBytes = vectorSize * sizeof(*input.data());

    float* dInput;
    float* dOutput;
    hipMalloc(&dInput, sizeInBytes);
    hipMalloc(&dOutput, sizeInBytes);

    hipMemcpy(dInput, input.data(), sizeInBytes, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, 0);
    size_t threadsBlock = deviceProps.maxThreadsPerBlock;
    size_t blocksGrid = (vectorSize + threadsBlock - 1) / threadsBlock;

    geluKernel<<<blocksGrid, threadsBlock>>>(dInput, dOutput, vectorSize);
    hipMemcpy(result.data(), dOutput, sizeInBytes, hipMemcpyDeviceToHost);

    hipFree(dInput);
    hipFree(dOutput);
    
    return result;
}